#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/native/quantized/fake_quant_affine.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <thrust/tuple.h>
#include <cmath>

/* Fake quantize a tensor
Args:
  output: output tensor.
  input : input tensor.
  sc:  scale to quantize the input tensor to
  zero_point: zero_point
  quant_min: minimum quantized value
  quant_max: maximum quantized value
Returns:
  Fake quantized tensor (float dtype).
*/
namespace at {
namespace native {
void fake_quantize_tensor_kernel_cuda(
    Tensor& output,
    const Tensor& input,
    float scale,
    int64_t zero_point,
    int64_t quant_min,
    int64_t quant_max) {
  // scalar type of this function is guaranteed to be float
  float inv_scale = 1.0f / scale;
  auto iter = TensorIteratorConfig()
    .check_all_same_dtype(false)
    .add_output(output)
    .add_input(input)
    .build();
  gpu_kernel(iter,
    [=] GPU_LAMBDA (float input_val) -> float {
      return (fminf(
                quant_max,
                fmaxf(
                    quant_min,
                    static_cast<int64_t>(std::nearbyint(
                        input_val * inv_scale + zero_point)))) -
            zero_point) *
          scale;
    });
}

void fake_quantize_grad_tensor_kernel_cuda(
    Tensor& input_grad,
    const Tensor& input,
    const Tensor& output_grad,
    float scale,
    int64_t zero_point,
    int64_t quant_min,
    int64_t quant_max) {
  // scalar type of this function is guaranteed to be float
  float inv_scale = 1.0f / scale;
  auto iter = TensorIteratorConfig()
    .check_all_same_dtype(false)
    .add_output(input_grad)
    .add_input(output_grad)
    .add_input(input)
    .build();
  gpu_kernel(iter,
    [=] GPU_LAMBDA (float dy, float x) -> float {
      int64_t Xq = std::nearbyint(x * inv_scale + zero_point);
      return (Xq >= quant_min && Xq <= quant_max) * dy;
    });
}

void _fake_quantize_grad_learnable_tensor_kernel_cuda(
    TensorIterator& iter,
    float scale,
    float inv_scale,
    int64_t zero_point,
    int64_t quant_min,
    int64_t quant_max) {
  float dscale_small = quant_min - zero_point;
  float dscale_big = quant_max - zero_point;
  gpu_kernel_multiple_outputs(
    iter, [=] GPU_LAMBDA (float XInput, float dYInput) -> thrust::tuple<float, float, float> {
      float dXOutput, dZeroPointOutput, dScaleOutput;
      int64_t xq = std::nearbyint(zero_point + XInput * inv_scale);
      dXOutput = dYInput * (xq >= quant_min && xq <= quant_max);
      xq = std::max(std::min(xq, quant_max), quant_min);
      float xfq = static_cast<float>((xq - zero_point) * scale);
      if (xq == quant_min || xq == quant_max) {
        dZeroPointOutput = (dYInput) * (-1) * scale;
        dScaleOutput = (xq == quant_min) ? (dYInput * dscale_small) : (dYInput * dscale_big);
      } else {
        dZeroPointOutput = 0;
        dScaleOutput = (dYInput) * (xfq - (XInput)) * inv_scale;
      }
      return {dXOutput, dScaleOutput, dZeroPointOutput};
  });
}

REGISTER_DISPATCH(fake_quant_tensor_stub, &fake_quantize_tensor_kernel_cuda);
REGISTER_DISPATCH(fake_quant_grad_tensor_stub, &fake_quantize_grad_tensor_kernel_cuda);
REGISTER_DISPATCH(fake_quant_grad_learnable_tensor_stub, &_fake_quantize_grad_learnable_tensor_kernel_cuda);

// Fake quantize per channel

void fake_quant_per_channel_cuda(TensorIterator &iter, int64_t quant_min, int64_t quant_max) {
  gpu_kernel(iter,
    [=] GPU_LAMBDA (float input_val, float scale, int64_t zero_point) -> float {
      float inv_scale = 1.0f / scale;
      return (fminf(
                quant_max,
                fmaxf(
                    quant_min,
                    static_cast<int64_t>(std::nearbyint(
                        input_val * inv_scale + zero_point)))) -
            zero_point) *
          scale;
    });
}

void fake_quant_grad_per_channel_cuda(TensorIterator &iter, int64_t quant_min, int64_t quant_max) {
  gpu_kernel(iter,
    [=] GPU_LAMBDA (float x, float dy, float scale, int64_t zero_point) -> float {
      float inv_scale = 1.0f / scale;
      int64_t Xq = std::nearbyint(x * inv_scale + zero_point);
      return (Xq >= quant_min && Xq <= quant_max) * dy;
    });
}

void _fake_quantize_grad_learnable_scale_channel_kernel_cuda(
    Tensor& input_grad,
    const Tensor& input,
    const Tensor& output_grad,
    float scale,
    int64_t zero_point,
    int64_t quant_min,
    int64_t quant_max) {
  // scalar type of this function is guaranteed to be float
  float inv_scale = 1.0f / scale;
  float grad_small = quant_min - zero_point;
  float grad_big = quant_max - zero_point;

  auto iter = TensorIterator::binary_op(input_grad, input, output_grad);
  gpu_kernel(iter,
    [=] GPU_LAMBDA (float x, float dy) -> float {
      int64_t xq = static_cast<int64_t>(zero_point + std::nearbyint(x * inv_scale));
      xq = std::max(std::min(xq, quant_max), quant_min);
      float x_fq = static_cast<float>((xq - zero_point) * scale);
      if (xq == quant_min) {
        return dy * grad_small;
      } else if (xq == quant_max) {
        return dy * grad_big;
      }
      return dy * (x_fq - x) * inv_scale;
    });
}

void _fake_quantize_grad_learnable_zero_point_channel_kernel_cuda(
    Tensor& input_grad,
    const Tensor& input,
    const Tensor& output_grad,
    float scale,
    int64_t zero_point,
    int64_t quant_min,
    int64_t quant_max) {
  // scalar type of this function is guaranteed to be float
  float inv_scale = 1.0f / scale;
  auto iter = TensorIterator::binary_op(input_grad, input, output_grad);
  gpu_kernel(iter,
    [=] GPU_LAMBDA (float x, float dy) -> float {
      int64_t xq = static_cast<int64_t>(zero_point + std::nearbyint(x * inv_scale));
      xq = std::max(std::min(xq, quant_max), quant_min);
      if (xq == quant_min || xq == quant_max) {
        return dy * (-1) * scale;
      }
      return 0;
    });
}

REGISTER_DISPATCH(fake_quant_per_channel_stub, &fake_quant_per_channel_cuda);
REGISTER_DISPATCH(fake_quant_grad_per_channel_stub, &fake_quant_grad_per_channel_cuda);
REGISTER_DISPATCH(fake_quant_grad_learnable_scale_channel_stub, &_fake_quantize_grad_learnable_scale_channel_kernel_cuda);
REGISTER_DISPATCH(fake_quant_grad_learnable_zero_point_channel_stub, &_fake_quantize_grad_learnable_zero_point_channel_kernel_cuda);

} // namespace native
} // namespace at
